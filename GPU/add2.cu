#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


#define KEY_LEN 20
//#define DATA_PATH "/home/zhanghw/train_new150w.txt"
//#define BATCH_SIZE (1024 * 256)

#define MIN(a,b) ((a)<(b)?(a):(b))

#define mix(a,b,c) \
{ \
  a -= b; a -= c; a ^= (c>>13); \
  b -= c; b -= a; b ^= (a<<8); \
  c -= a; c -= b; c ^= (b>>13); \
  a -= b; a -= c; a ^= (c>>12);  \
  b -= c; b -= a; b ^= (a<<16); \
  c -= a; c -= b; c ^= (b>>5); \
  a -= b; a -= c; a ^= (c>>3);  \
  b -= c; b -= a; b ^= (a<<10); \
  c -= a; c -= b; c ^= (b>>15); \
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
      fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
      assert(result == hipSuccess);
    }
#endif
    return result;
}

int read_data(int** config, int** onchip, char** offchip, char** stash, int* malloc_size, char** key_set, int** offset) {
    FILE * fp = fopen(DATA_PATH, "r");
    int list_num;
    fscanf(fp, "%d", &list_num);
    malloc_size[0] = sizeof(int) * (list_num + 4);
    *config = (int*)malloc(sizeof(int) * (list_num + 4));
    (*config)[0] = list_num;
    for (int i = 1; i <= list_num + 3; ++i)
        fscanf(fp, "%d", (*config) + i);
    int stash_size;
    fscanf(fp, "%d", &stash_size);
    malloc_size[1] = (stash_size + 1) * KEY_LEN * sizeof(char);
    *stash = (char*)malloc((stash_size + 1) * KEY_LEN * sizeof(char));
    char* stash_p = *stash;
    for (int i = 0; i < stash_size; ++i) {
        fscanf(fp, "%s", stash_p);
        stash_p += KEY_LEN;
    }
    *stash_p = 0;
    int tot_size = 0;
    for (int i = 1; i <= list_num; ++i)
        tot_size += (*config)[i];
    int bucket_num = (*config)[list_num + 1];
    malloc_size[2] = tot_size * bucket_num * sizeof(int);
    *onchip = (int*)malloc(tot_size * bucket_num * sizeof(int));
    memset(*onchip, -1, tot_size * bucket_num * sizeof(int));
    *offchip = (char*)malloc(tot_size * bucket_num * KEY_LEN * sizeof(char));
    malloc_size[3] = sizeof(int) * list_num;
    *offset = (int*)malloc(sizeof(int) * list_num);
    malloc_size[4] = bucket_num * list_num;
    (*offset)[0] = 0;
    for (int i = 0; i < list_num-1; ++i) {
        (*offset)[i+1] = (*offset)[i] + (*config)[i + 1] * bucket_num;
    }
    int data_size;
    fscanf(fp, "%d", &data_size);
    int key_num = stash_size + data_size;
    checkCuda( hipHostMalloc((void**)key_set, key_num * KEY_LEN * sizeof(char)) );
    //*key_set = (char*)malloc(key_num * KEY_LEN * sizeof(char));
    memcpy(*key_set, *stash, stash_size * KEY_LEN * sizeof(char));
    char* key_set_p = *key_set + stash_size * KEY_LEN;
    while (data_size--) {
        char str[KEY_LEN];
        int list_pos, bucket_pos, slot_pos, finger_print, hash_id;
        fscanf(fp, "%s %d %d %d %d %d", str, &list_pos, &bucket_pos, &slot_pos, &finger_print, &hash_id);
        strcpy(key_set_p, str);
        key_set_p += KEY_LEN;
        (*onchip)[(*offset)[list_pos] + bucket_pos * bucket_num + slot_pos] = ((finger_print << 16) | (hash_id & 0xffff));
        strcpy((*offchip)+((*offset)[list_pos]+bucket_pos*bucket_num+slot_pos)*KEY_LEN, str);
    }
    fclose(fp);
    return key_num;
}


__constant__ 
unsigned int prime32[1229] = {
    2, 3, 5, 7, 11, 13, 17, 19, 23, 29,
    31, 37, 41, 43, 47, 53, 59, 61, 67, 71,
    73, 79, 83, 89, 97, 101, 103, 107, 109, 113,
    127, 131, 137, 139, 149, 151, 157, 163, 167, 173,
    179, 181, 191, 193, 197, 199, 211, 223, 227, 229,
    233, 239, 241, 251, 257, 263, 269, 271, 277, 281,
    283, 293, 307, 311, 313, 317, 331, 337, 347, 349,
    353, 359, 367, 373, 379, 383, 389, 397, 401, 409,
    419, 421, 431, 433, 439, 443, 449, 457, 461, 463,
    467, 479, 487, 491, 499, 503, 509, 521, 523, 541,
    547, 557, 563, 569, 571, 577, 587, 593, 599, 601,
    607, 613, 617, 619, 631, 641, 643, 647, 653, 659,
    661, 673, 677, 683, 691, 701, 709, 719, 727, 733,
    739, 743, 751, 757, 761, 769, 773, 787, 797, 809,
    811, 821, 823, 827, 829, 839, 853, 857, 859, 863,
    877, 881, 883, 887, 907, 911, 919, 929, 937, 941,
    947, 953, 967, 971, 977, 983, 991, 997,
    1009, 1013, 1019, 1021, 1031, 1033, 1039, 1049, 1051, 1061,
    1063, 1069, 1087, 1091, 1093, 1097, 1103, 1109, 1117, 1123,
    1129, 1151, 1153, 1163, 1171, 1181, 1187, 1193, 1201, 1213,
    1217, 1223, 1229, 1231, 1237, 1249, 1259, 1277, 1279, 1283,
    1289, 1291, 1297, 1301, 1303, 1307, 1319, 1321, 1327, 1361,
    1367, 1373, 1381, 1399, 1409, 1423, 1427, 1429, 1433, 1439,
    1447, 1451, 1453, 1459, 1471, 1481, 1483, 1487, 1489, 1493,
    1499, 1511, 1523, 1531, 1543, 1549, 1553, 1559, 1567, 1571,
    1579, 1583, 1597, 1601, 1607, 1609, 1613, 1619, 1621, 1627,
    1637, 1657, 1663, 1667, 1669, 1693, 1697, 1699, 1709, 1721,
    1723, 1733, 1741, 1747, 1753, 1759, 1777, 1783, 1787, 1789,
    1801, 1811, 1823, 1831, 1847, 1861, 1867, 1871, 1873, 1877,
    1879, 1889, 1901, 1907, 1913, 1931, 1933, 1949, 1951, 1973,
    1979, 1987, 1993, 1997, 1999, 2003, 2011, 2017, 2027, 2029,
    2039, 2053, 2063, 2069, 2081, 2083, 2087, 2089, 2099, 2111,
    2113, 2129, 2131, 2137, 2141, 2143, 2153, 2161, 2179, 2203,
    2207, 2213, 2221, 2237, 2239, 2243, 2251, 2267, 2269, 2273,
    2281, 2287, 2293, 2297, 2309, 2311, 2333, 2339, 2341, 2347,
    2351, 2357, 2371, 2377, 2381, 2383, 2389, 2393, 2399, 2411,
    2417, 2423, 2437, 2441, 2447, 2459, 2467, 2473, 2477, 2503,
    2521, 2531, 2539, 2543, 2549, 2551, 2557, 2579, 2591, 2593,
    2609, 2617, 2621, 2633, 2647, 2657, 2659, 2663, 2671, 2677,
    2683, 2687, 2689, 2693, 2699, 2707, 2711, 2713, 2719, 2729,
    2731, 2741, 2749, 2753, 2767, 2777, 2789, 2791, 2797, 2801,
    2803, 2819, 2833, 2837, 2843, 2851, 2857, 2861, 2879, 2887,
    2897, 2903, 2909, 2917, 2927, 2939, 2953, 2957, 2963, 2969,
    2971, 2999, 3001, 3011, 3019, 3023, 3037, 3041, 3049, 3061,
    3067, 3079, 3083, 3089, 3109, 3119, 3121, 3137, 3163, 3167,
    3169, 3181, 3187, 3191, 3203, 3209, 3217, 3221, 3229, 3251,
    3253, 3257, 3259, 3271, 3299, 3301, 3307, 3313, 3319, 3323,
    3329, 3331, 3343, 3347, 3359, 3361, 3371, 3373, 3389, 3391,
    3407, 3413, 3433, 3449, 3457, 3461, 3463, 3467, 3469, 3491,
    3499, 3511, 3517, 3527, 3529, 3533, 3539, 3541, 3547, 3557,
    3559, 3571, 3581, 3583, 3593, 3607, 3613, 3617, 3623, 3631,
    3637, 3643, 3659, 3671, 3673, 3677, 3691, 3697, 3701, 3709,
    3719, 3727, 3733, 3739, 3761, 3767, 3769, 3779, 3793, 3797,
    3803, 3821, 3823, 3833, 3847, 3851, 3853, 3863, 3877, 3881,
    3889, 3907, 3911, 3917, 3919, 3923, 3929, 3931, 3943, 3947,
    3967, 3989, 4001, 4003, 4007, 4013, 4019, 4021, 4027, 4049,
    4051, 4057, 4073, 4079, 4091, 4093, 4099, 4111, 4127, 4129,
    4133, 4139, 4153, 4157, 4159, 4177, 4201, 4211, 4217, 4219,
    4229, 4231, 4241, 4243, 4253, 4259, 4261, 4271, 4273, 4283,
    4289, 4297, 4327, 4337, 4339, 4349, 4357, 4363, 4373, 4391,
    4397, 4409, 4421, 4423, 4441, 4447, 4451, 4457, 4463, 4481,
    4483, 4493, 4507, 4513, 4517, 4519, 4523, 4547, 4549, 4561,
    4567, 4583, 4591, 4597, 4603, 4621, 4637, 4639, 4643, 4649,
    4651, 4657, 4663, 4673, 4679, 4691, 4703, 4721, 4723, 4729,
    4733, 4751, 4759, 4783, 4787, 4789, 4793, 4799, 4801, 4813,
    4817, 4831, 4861, 4871, 4877, 4889, 4903, 4909, 4919, 4931,
    4933, 4937, 4943, 4951, 4957, 4967, 4969, 4973, 4987, 4993,
    4999, 5003, 5009, 5011, 5021, 5023, 5039, 5051, 5059, 5077,
    5081, 5087, 5099, 5101, 5107, 5113, 5119, 5147, 5153, 5167,
    5171, 5179, 5189, 5197, 5209, 5227, 5231, 5233, 5237, 5261,
    5273, 5279, 5281, 5297, 5303, 5309, 5323, 5333, 5347, 5351,
    5381, 5387, 5393, 5399, 5407, 5413, 5417, 5419, 5431, 5437,
    5441, 5443, 5449, 5471, 5477, 5479, 5483, 5501, 5503, 5507,
    5519, 5521, 5527, 5531, 5557, 5563, 5569, 5573, 5581, 5591,
    5623, 5639, 5641, 5647, 5651, 5653, 5657, 5659, 5669, 5683,
    5689, 5693, 5701, 5711, 5717, 5737, 5741, 5743, 5749, 5779,
    5783, 5791, 5801, 5807, 5813, 5821, 5827, 5839, 5843, 5849,
    5851, 5857, 5861, 5867, 5869, 5879, 5881, 5897, 5903, 5923,
    5927, 5939, 5953, 5981, 5987, 6007, 6011, 6029, 6037, 6043,
    6047, 6053, 6067, 6073, 6079, 6089, 6091, 6101, 6113, 6121,
    6131, 6133, 6143, 6151, 6163, 6173, 6197, 6199, 6203, 6211,
    6217, 6221, 6229, 6247, 6257, 6263, 6269, 6271, 6277, 6287,
    6299, 6301, 6311, 6317, 6323, 6329, 6337, 6343, 6353, 6359,
    6361, 6367, 6373, 6379, 6389, 6397, 6421, 6427, 6449, 6451,
    6469, 6473, 6481, 6491, 6521, 6529, 6547, 6551, 6553, 6563,
    6569, 6571, 6577, 6581, 6599, 6607, 6619, 6637, 6653, 6659,
    6661, 6673, 6679, 6689, 6691, 6701, 6703, 6709, 6719, 6733,
    6737, 6761, 6763, 6779, 6781, 6791, 6793, 6803, 6823, 6827,
    6829, 6833, 6841, 6857, 6863, 6869, 6871, 6883, 6899, 6907,
    6911, 6917, 6947, 6949, 6959, 6961, 6967, 6971, 6977, 6983,
    6991, 6997, 7001, 7013, 7019, 7027, 7039, 7043, 7057, 7069,
    7079, 7103, 7109, 7121, 7127, 7129, 7151, 7159, 7177, 7187,
    7193, 7207, 7211, 7213, 7219, 7229, 7237, 7243, 7247, 7253,
    7283, 7297, 7307, 7309, 7321, 7331, 7333, 7349, 7351, 7369,
    7393, 7411, 7417, 7433, 7451, 7457, 7459, 7477, 7481, 7487,
    7489, 7499, 7507, 7517, 7523, 7529, 7537, 7541, 7547, 7549,
    7559, 7561, 7573, 7577, 7583, 7589, 7591, 7603, 7607, 7621,
    7639, 7643, 7649, 7669, 7673, 7681, 7687, 7691, 7699, 7703,
    7717, 7723, 7727, 7741, 7753, 7757, 7759, 7789, 7793, 7817,
    7823, 7829, 7841, 7853, 7867, 7873, 7877, 7879, 7883, 7901,
    7907, 7919, 7927, 7933, 7937, 7949, 7951, 7963, 7993, 8009,
    8011, 8017, 8039, 8053, 8059, 8069, 8081, 8087, 8089, 8093,
    8101, 8111, 8117, 8123, 8147, 8161, 8167, 8171, 8179, 8191,
    8209, 8219, 8221, 8231, 8233, 8237, 8243, 8263, 8269, 8273,
    8287, 8291, 8293, 8297, 8311, 8317, 8329, 8353, 8363, 8369,
    8377, 8387, 8389, 8419, 8423, 8429, 8431, 8443, 8447, 8461,
    8467, 8501, 8513, 8521, 8527, 8537, 8539, 8543, 8563, 8573,
    8581, 8597, 8599, 8609, 8623, 8627, 8629, 8641, 8647, 8663,
    8669, 8677, 8681, 8689, 8693, 8699, 8707, 8713, 8719, 8731,
    8737, 8741, 8747, 8753, 8761, 8779, 8783, 8803, 8807, 8819,
    8821, 8831, 8837, 8839, 8849, 8861, 8863, 8867, 8887, 8893,
    8923, 8929, 8933, 8941, 8951, 8963, 8969, 8971, 8999, 9001,
    9007, 9011, 9013, 9029, 9041, 9043, 9049, 9059, 9067, 9091,
    9103, 9109, 9127, 9133, 9137, 9151, 9157, 9161, 9173, 9181,
    9187, 9199, 9203, 9209, 9221, 9227, 9239, 9241, 9257, 9277,
    9281, 9283, 9293, 9311, 9319, 9323, 9337, 9341, 9343, 9349,
    9371, 9377, 9391, 9397, 9403, 9413, 9419, 9421, 9431, 9433,
    9437, 9439, 9461, 9463, 9467, 9473, 9479, 9491, 9497, 9511,
    9521, 9533, 9539, 9547, 9551, 9587, 9601, 9613, 9619, 9623,
    9629, 9631, 9643, 9649, 9661, 9677, 9679, 9689, 9697, 9719,
    9721, 9733, 9739, 9743, 9749, 9767, 9769, 9781, 9787, 9791,
    9803, 9811, 9817, 9829, 9833, 9839, 9851, 9857, 9859, 9871,
    9883, 9887, 9901, 9907, 9923, 9929, 9931, 9941, 9949, 9967,
    9973
};

__device__
inline
int mystrlen(const char* str) {
    int len = 0;
    while (*(str++)) ++len;
    return len;
}

__device__
inline
unsigned int
hash(unsigned int initval, const char* str)
{
    unsigned int len = mystrlen(str);
    initval = prime32[initval%1229];
    //register ub4 a,b,c,len;
    unsigned int a,b,c;
    /* Set up the internal state */
    //len = length;
    a = b = 0x9e3779b9;  /* the golden ratio; an arbitrary value */
    c = initval;         /* the previous hash value */

    /*---------------------------------------- handle most of the key */
    while (len >= 12)
    {
        a += (str[0] +((unsigned int)str[1]<<8) +((unsigned int)str[2]<<16) +((unsigned int)str[3]<<24));
        b += (str[4] +((unsigned int)str[5]<<8) +((unsigned int)str[6]<<16) +((unsigned int)str[7]<<24));
        c += (str[8] +((unsigned int)str[9]<<8) +((unsigned int)str[10]<<16)+((unsigned int)str[11]<<24));
        mix(a,b,c);
        str += 12; len -= 12;
    }

    /*------------------------------------- handle the last 11 bytes */
    c += len;
    switch(len)              /* all the case statements fall through */
    {
        case 11: c+=((unsigned int)str[10]<<24);
        case 10: c+=((unsigned int)str[9]<<16);
        case 9 : c+=((unsigned int)str[8]<<8);
        /* the first byte of c is reserved for the length */
        case 8 : b+=((unsigned int)str[7]<<24);
        case 7 : b+=((unsigned int)str[6]<<16);
        case 6 : b+=((unsigned int)str[5]<<8);
        case 5 : b+=str[4];
        case 4 : a+=((unsigned int)str[3]<<24);
        case 3 : a+=((unsigned int)str[2]<<16);
        case 2 : a+=((unsigned int)str[1]<<8);
        case 1 : a+=str[0];
        /* case 0: nothing left to add */
    }
    mix(a,b,c);
    /*-------------------------------------------- report the result */
    return c;
}

__device__
inline
void decode(
    const int* config,
    int* list_num,
    int* bucket_num,
    int* mask1,
    int* mask2)
{
    *list_num = config[0];
    *bucket_num = config[*list_num + 1];
    *mask1 = config[*list_num + 2];
    *mask2 = config[*list_num + 3];
}

__device__
inline
int equal(const char* s1, const char* s2, int len) {
    while (len--) {
        if (*(s1++) != *(s2++)) {
            return 0;
        }
    }
    return 1;
}

__device__
inline
int lookup_stash(const char* stash, const char* key, const int len) {
    for (; *stash; stash += KEY_LEN) {
        if (equal(stash, key, len)) {
            return 1;
        }
    }
    return 0;
}

__global__
void lookup_key(const int key_num, const int h_offset, const char* key_set, const int* onchip, const int* config, const char* stash, const int* list_offset, int* res_tmp) {
    int list_num, bucket_num, mask1, mask2;
    decode(config, &list_num, &bucket_num, &mask1, &mask2);
    int id = h_offset * list_num * bucket_num + threadIdx.x + blockIdx.x * blockDim.x;
    int key_idx = id / (list_num * bucket_num), hash_idx = id % (list_num * bucket_num);
    
    if (key_idx >= key_num) return;
    int list_idx = hash_idx / bucket_num, slot_idx = hash_idx % bucket_num;
    const char* key = key_set + key_idx * KEY_LEN;    
    int key_len = mystrlen(key);
    if (lookup_stash(stash, key, key_len)) {
        res_tmp[id] = -1;
        return;
    }
    int hash_pos, list_num_half = list_num / 2;
    int hash_offset = (list_idx < list_num_half ? list_idx : (list_idx - list_num_half));
    unsigned int hash_val = hash(1000 + hash_offset, key);
    if (list_idx < list_num_half) {
        hash_pos = hash_val % (unsigned)config[list_idx + 1];
    } else {
        hash_pos = (hash_val % (unsigned)config[1 + list_idx - list_num_half]) / (config[1 + list_idx - list_num_half] / config[1 + list_idx]);
    }
    const int* bucket = onchip + list_offset[list_idx] + hash_pos * bucket_num;
    int mask = (list_idx < list_num_half ? mask1 : mask2);
    int fingerid = bucket[slot_idx] & 0xffff;
    if (fingerid == 65535) {
        res_tmp[id] = -2;
        return;
    }
    int fingerprint = (bucket[slot_idx] >> 16) & 0xffff;
    //printf("%s %d %d %d %d\n", key, fingerid, mask, fingerprint, hash(1000 + fingerid, key) % (unsigned)mask);
    if (fingerprint == hash(1000 + fingerid, key) % (unsigned)mask) {
        res_tmp[id] = (list_offset[list_idx] + hash_pos * bucket_num + slot_idx) * KEY_LEN;
        return;
    }
    res_tmp[id] = -2;
}

__global__
void collect_res(const int key_num, const int h_offset, const int* res_tmp, int* result, const int check_num) {
    int id = h_offset + threadIdx.x + blockIdx.x * blockDim.x;
    int res = -3;

    for (int i = 0; i < check_num; ++i) {
        int val = res_tmp[id * check_num + i];
        if (val == -1) {
            result[id] = -1;
            return;
        } else if (val >= 0) {
            if (res == -3) res = val;
            else {
                result[id] = -2;
                return;
            }
        }
    }
    result[id] = res;
    return;
}

void device_setup(int* malloc_size, int* config, int key_num, int** d_config, int** d_onchip, int** d_result,char** d_stash, char** d_key_set, int** d_res_tmp, int** d_list_offset) {
    checkCuda( hipMalloc((void**)d_config, malloc_size[0]) );
    checkCuda( hipMalloc((void**)d_stash, malloc_size[1]) );
    checkCuda( hipMalloc((void**)d_onchip, malloc_size[2]) );
    checkCuda( hipMalloc((void**)d_list_offset, malloc_size[3]) );
    checkCuda( hipMalloc((void**)d_key_set, key_num * KEY_LEN * sizeof(char)) );
    checkCuda( hipMalloc((void**)d_result, key_num * sizeof(int)) );
    checkCuda( hipMalloc((void**)d_res_tmp, key_num * malloc_size[4] * sizeof(int)) );
}

int main() {
    int *config, *onchip, malloc_size[5], *result, *list_offset;
    int *d_config, *d_onchip, *d_result, *d_list_offset, *d_res_tmp;
    char *offchip, *stash, *key_set;
    char *d_stash, *d_key_set;
    int key_num = read_data(&config, &onchip, &offchip, &stash, malloc_size, &key_set, &list_offset);
    checkCuda( hipHostMalloc((void**)&result, sizeof(int) * key_num) );
    device_setup(malloc_size, config, key_num, &d_config, &d_onchip, &d_result, &d_stash, &d_key_set, &d_res_tmp, &d_list_offset);
    
    checkCuda( hipMemcpy(d_config, config, malloc_size[0], hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_stash, stash, malloc_size[1], hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_onchip, onchip, malloc_size[2], hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_list_offset, list_offset, malloc_size[3], hipMemcpyHostToDevice) );

    const int blockSize = 256;
    int batch_num = (key_num + BATCH_SIZE - 1) / BATCH_SIZE, key_num_left = key_num;

    hipEvent_t startEvent, stopEvent;
    hipStream_t stream[batch_num];
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );
    for (int i = 0; i < batch_num; ++i)
        checkCuda( hipStreamCreate(&stream[i]) );

    checkCuda( hipEventRecord(startEvent, 0) );
    for (int i = 0; i < batch_num; ++i, key_num_left -= BATCH_SIZE) {
        int real_size = MIN(BATCH_SIZE, key_num_left);
        int offset = i * BATCH_SIZE;
        checkCuda( hipMemcpyAsync(&d_key_set[offset * KEY_LEN], &key_set[offset * KEY_LEN], real_size * KEY_LEN * sizeof(char), hipMemcpyHostToDevice, stream[i]) );
        lookup_key<<<(real_size * malloc_size[4] + blockSize - 1) / blockSize, blockSize, 0, stream[i]>>>(key_num, offset, d_key_set, d_onchip, d_config, d_stash, d_list_offset, d_res_tmp);
        collect_res<<<(real_size + blockSize - 1) / blockSize, blockSize, 0, stream[i]>>>(key_num, offset, d_res_tmp, d_result, malloc_size[4]);
        checkCuda( hipMemcpyAsync(&result[offset], &d_result[offset], real_size * sizeof(int), hipMemcpyDeviceToHost, stream[i]) );
    }

    key_num_left = key_num;
    for (int i = 0; i < batch_num; ++i, key_num_left -= BATCH_SIZE) {
        int real_size = MIN(BATCH_SIZE, key_num_left);
        int offset = i * BATCH_SIZE;
        checkCuda( hipStreamSynchronize(stream[i]) );

        for (int j = offset; j < offset + real_size; ++j)
            if (result[j] >= 0)
                if (strcmp(offchip + result[j], key_set + j * KEY_LEN))
                    printf("?\n");
    }
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    float ms;
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%f Mqps\n", key_num / ms / 1000);

    free(list_offset);
    free(config);
    free(onchip);
    hipHostFree(result);
    free(offchip);
    free(stash);
    hipHostFree(key_set);

    hipFree(d_list_offset);
    hipFree(d_res_tmp);
    hipFree(d_config);
    hipFree(d_onchip);
    hipFree(d_result);
    hipFree(d_stash);
    hipFree(d_key_set);
}